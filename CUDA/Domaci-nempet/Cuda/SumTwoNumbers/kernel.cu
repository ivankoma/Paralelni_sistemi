#include "hip/hip_runtime.h"
#include <iostream>

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

using namespace std;

__global__ void add(int *a, int *b, int *sum)
{
	*sum = *a + *b;
}

__global__ void add2(int a, int b, int *sum)
{
	*sum = *sum + a + b;
}

int main()
{
	int a = 100;
	int b = 99;
	int sum;
	int *d_sum;
	int *d_a, *d_b;

	// ------ sa pointerima ----------

	hipMalloc(&d_sum, sizeof(int));
	hipMalloc(&d_a, sizeof(int));
	hipMalloc(&d_b, sizeof(int));

	hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

	add<<<1,1>>> (d_a, d_b, d_sum);

	hipMemcpy(&sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);

	cout << sum << endl;

	hipFree(d_sum);
	hipFree(d_a);
	hipFree(d_b);

	// -------------------------

	// ------- po vrednosti ---------------

	hipMalloc(&d_sum, sizeof(int));

	hipMemcpy(d_sum, &sum, sizeof(int), hipMemcpyHostToDevice);

	add2 << <1, 1 >> > (a, b, d_sum);

	hipMemcpy(&sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);

	cout << sum << endl;

	hipFree(d_sum);

	// ----------------------------

	hipDeviceReset();

	cout << endl;
	system("pause");
    return 0;
}