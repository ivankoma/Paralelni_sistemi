#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#define size 128
#define threads 32

using namespace std;

__global__ void callOperation(int *niz, int *res, int k, int n)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	if (tid >= n) {
		return;
	}

	if (niz[tid] == k) {
		atomicAdd(res, 1);
	}
}

int main()
{
	int k = 128;
	int res = 0;
	int *niz;
	int *d_res, *d_niz;

	niz = (int*)malloc(size * sizeof(int));

	for (int i = 0; i < size; i++) {
		niz[i] = size;
	}

	hipMalloc((void**)&d_niz, size * sizeof(int));
	hipMalloc((void**)&d_res, sizeof(int));

	hipMemcpy(d_niz, niz, size * sizeof(int), hipMemcpyHostToDevice);

	dim3 numberOfBlocks(size / threads + 1, 1, 1);
	dim3 numberOfThreads(threads, 1, 1);

	callOperation << <numberOfBlocks, numberOfThreads >> > (d_niz, d_res, k, size);

	hipMemcpy(&res, d_res, sizeof(int), hipMemcpyDeviceToHost);

	cout << "Rezultat je: " << res << endl;

	hipFree(d_res);
	hipFree(d_niz);
	free(niz);

	system("PAUSE");
	return 0;
}