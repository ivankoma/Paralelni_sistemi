#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#define size 16
#define threads 32

using namespace std;


__global__ void callOperationMin(int * niz)
{
	int tid = threadIdx.x;
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;

	for (unsigned int offset = blockDim.x / 2; offset > 0; offset >>= 1)
	{
		if (tid < offset)
		{
			if (niz[tid] > niz[tid + offset])
			{
				niz[tid] = niz[tid + offset];
			}
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		niz[blockIdx.x] = niz[tid];
	}
}

__global__ void callOperationMax(int * niz)
{
	int tid = threadIdx.x;
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;

	for (unsigned int offset = blockDim.x / 2; offset > 0; offset >>= 1)
	{
		if(tid < offset)
		{
			if (niz[tid] < niz[tid + offset])
			{
				niz[tid] = niz[tid + offset];
			}

			__syncthreads();
		}
	}

	if (tid == 0)
	{
		niz[blockIdx.x] = niz[tid];
	}
}

int main()
{
	int *niz;
	int *d_niz;

	niz = (int*)malloc(size * sizeof(int));
	int * niz2 = (int*)malloc(size * sizeof(int));

	for (int i = 0; i < size; i++) {
		niz[i] = i;
	}

	cout << "Niz: " << endl;
	for (int i = 0; i < size; i++)
	{
		cout << niz[i] << "\t";
	}

	hipMalloc((void**)&d_niz, size * sizeof(int));
	hipMemcpy(d_niz, niz, size * sizeof(int), hipMemcpyHostToDevice);

	callOperationMax << <size / threads + 1, threads >> > (d_niz);
	int max;
	hipMemcpy(&max, d_niz, sizeof(int), hipMemcpyDeviceToHost);

	/*
	callOperationMin << <size / threads + 1, threads >> > (d_niz);
	int min;
	hipMemcpy(&min, d_niz, sizeof(int), hipMemcpyDeviceToHost);
	*/

	hipMemcpy(niz2, d_niz, size * sizeof(int), hipMemcpyDeviceToHost);

	cout << "\n\nMax je: " << max;
	//cout << "\n\nMin je: " << min;

	cout << "\n\nNiz: " << endl;
	for (int i = 0; i < size; i++)
	{
		cout << niz2[i] << "\t";
	}

	hipFree(d_niz);
	free(niz);
	hipDeviceReset();

	cout << endl;
	system("PAUSE");
    return 0;
}
