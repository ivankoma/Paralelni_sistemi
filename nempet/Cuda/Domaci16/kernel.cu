#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>

#define size 5
#define threads 32

using namespace std;

__global__ void callOperation(int * a, int *b, int *res, int n)
{
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	int tidy = blockDim.y * blockIdx.y + threadIdx.y;

	if (tidx >= n || tidy >= n)
	{
		return;
	}
	int tid = tidx * n + tidy;

	res[tid] = a[tid] - b[tid];
	if (res[tid] < 0)
	{
		res[tid] = 0;
	}
}

__global__ void callOperationSharedStatic(int * a, int *b, int *res, int n)
{
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	int tidy = blockDim.y * blockIdx.y + threadIdx.y;

	if (tidx >= n || tidy >= n)
	{
		return;
	}
	int tid = tidx * n + tidy;

	__shared__ int s_a[size * size], s_b[size * size], s_res[size * size];

	s_a[tid] = a[tid];
	s_b[tid] = b[tid];

	s_res[tid] = s_a[tid] - s_b[tid];
	if (s_res[tid] < 0)
	{
		s_res[tid] = 0;
	}
	res[tid] = s_res[tid];
}

__global__ void callOperationSharedDynamic(int * a, int *b, int *res, int n)
{
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	int tidy = blockDim.y * blockIdx.y + threadIdx.y;

	if (tidx >= n || tidy >= n)
	{
		return;
	}
	int tid = tidx * n + tidy;

	extern __shared__ int data[];

	int *s_a = data;
	int *s_b = &s_a[size * size];
	int *s_res = &s_b[size * size];

	s_a[tid] = a[tid];
	s_b[tid] = b[tid];

	s_res[tid] = s_a[tid] - s_b[tid];
	if (s_res[tid] < 0)
	{
		s_res[tid] = 0;
	}
	res[tid] = s_res[tid];
}



int main()
{
	int *a, *b, *res;
	int *d_a, *d_b, *d_res;

	a = (int*)malloc(size * size * sizeof(int));
	b = (int*)malloc(size * size * sizeof(int));
	res = (int*)malloc(size * size * sizeof(int));

	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			a[i * size + j] = -i;
			b[i * size + j] = j;
		}
	}

	cout << "\nMat A:" << endl;
	for (int i = 0; i < size; i++)
	{
		cout << "\n";
		for (int j = 0; j < size; j++)
		{
			cout << a[i * size + j] << "\t";
		}
	}

	cout << "\nMat B:" << endl;
	for (int i = 0; i < size; i++)
	{
		cout << "\n";
		for (int j = 0; j < size; j++)
		{
			cout << b[i * size + j] << "\t";
		}
	}

	hipMalloc((void**)&d_a, size * size * sizeof(int));
	hipMalloc((void**)&d_b, size * size * sizeof(int));
	hipMalloc((void**)&d_res, size * size * sizeof(int));

	hipMemcpy(d_a, a, size * size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size * size * sizeof(int), hipMemcpyHostToDevice);

	dim3 numberOfBlocks(size / threads + 1, size / threads + 1, 1);
	dim3 numberOfThreads(threads, threads, 1);

	//callOperation << < numberOfBlocks, numberOfThreads >> > (d_a, d_b, d_res, size);
	//callOperationSharedStatic << < numberOfBlocks, numberOfThreads >> > (d_a, d_b, d_res, size);
	callOperationSharedDynamic << < numberOfBlocks, numberOfThreads, size * size * sizeof(int) + size * size * sizeof(int)+ size * size * sizeof(int) >> > (d_a, d_b, d_res, size);

	hipMemcpy(res, d_res, size * size * sizeof(int), hipMemcpyDeviceToHost);

	cout << "\nMat Res:" << endl;
	for (int i = 0; i < size; i++)
	{
		cout << "\n";
		for (int j = 0; j < size; j++)
		{
			cout << res[i * size + j] << "\t";
		}
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_res);
	free(a);
	free(b);
	free(res);
	hipDeviceReset();


	system("PAUSE");
	return 0;
}