#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#define size 5
#define threads 32

using namespace std;

__global__ void calculateMatrixFormula(int *a, int *b, int *res, int n)
{
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	int tidy = blockDim.y * blockIdx.y + threadIdx.y;

	if (tidx >= n || tidy >= n) {
		return;
	}

	int tid = tidx * n + tidy;
	res[tid] = a[tid] - b[tid];
}

__global__ void calculateMatrixFormulaSharedStatic(int *a, int *b, int *res, int n)
{
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	int tidy = blockDim.y * blockIdx.y + threadIdx.y;

	if (tidx >= n || tidy >= n) {
		return;
	}

	int tid = tidx * n + tidy;

	__shared__ int s_a[size * size], s_b[size * size], s_res[size * size];

	s_a[tid] = a[tid];
	s_b[tid] = b[tid];

	s_res[tid] = s_a[tid] - s_b[tid];
	res[tid] = s_res[tid];
}

__global__ void calculateMatrixFormulaSharedDynamic(int *a, int *b, int *res, int n)
{
	int tidx = blockDim.x * blockIdx.x + threadIdx.x;
	int tidy = blockDim.y * blockIdx.y + threadIdx.y;

	if (tidx >= n || tidy >= n) {
		return;
	}

	int tid = tidx * n + tidy;

	extern __shared__ int arrays[];

	int *s_a = arrays;
	int *s_b = &arrays[size * size];
	int *s_res = &s_b[size * size];

	s_a[tid] = a[tid];
	s_b[tid] = b[tid];

	s_res[tid] = s_a[tid] - s_b[tid];
	res[tid] = s_res[tid];
}

int main()
{
	int *a, *b, *res;
	int *d_a, *d_b, *d_res;

	a = (int*)malloc(size * size * sizeof(int));
	b = (int*)malloc(size * size * sizeof(int));
	res = (int*)malloc(size * size * sizeof(int));

	for (int i = 0; i < size; i++) {
		for (int j = 0; j < size; j++) {
			a[i * size + j] = i * size;
			b[i * size + j] = -i;
		}
	}

	cout << "\n\nVelicina matrice:" << size << endl;

	cout << "\n\nStampa matrice A:" << endl;
	for (int i = 0; i < size; i++) {
		cout << "\n";
		for (int j = 0; j < size; j++) {
			cout << a[i * size + j] << "\t";
		}
	}

	cout << "\n\nStampa matrice B:" << endl;
	for (int i = 0; i < size; i++) {
		cout << "\n";
		for (int j = 0; j < size; j++) {
			cout << b[i * size + j] << "\t";
		}
	}

	hipMalloc((void**)&d_a, size * size * sizeof(int));
	hipMalloc((void**)&d_b, size * size * sizeof(int));
	hipMalloc((void**)&d_res, size * size * sizeof(int));

	hipMemcpy(d_a, a, size * size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size * size * sizeof(int), hipMemcpyHostToDevice);

	dim3 numOfBlocks(size / threads + 1, size / threads + 1, 1);
	dim3 numOfThreads(threads, threads, 1);

	//calculateMatrixFormula << < numOfBlocks, numOfThreads >> > (d_a, d_b, d_res, size);
	//calculateMatrixFormulaSharedStatic << < numOfBlocks, numOfThreads >> > (d_a, d_b, d_res, size);
	calculateMatrixFormulaSharedDynamic << < numOfBlocks, numOfThreads, size * size * sizeof(int) + size * size * sizeof(int) + size * size * sizeof(int) >> > (d_a, d_b, d_res, size);

	hipMemcpy(res, d_res, size * size * sizeof(int), hipMemcpyDeviceToHost);

	cout << "\n\nStampa matrice A-B:" << endl;
	for (int i = 0; i < size; i++) {
		cout << "\n";
		for (int j = 0; j < size; j++) {
			cout << res[i * size + j] << "\t";
		}
	}

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_res);
	free(a);
	free(b);
	free(res);

	cout << endl;
	system("PAUSE");
	return 0;
}